#include "hip/hip_runtime.h"
#include <iostream>
#include <cstddef>

#include <boost/gil/image.hpp>
#include <boost/gil/image_view.hpp>
#include <boost/gil/typedefs.hpp>

namespace gil = boost::gil;

// namespace cuda
// {
// // view_type_from_pixel<Pixel, IsPlanar = false>::type
// // iterator_type_from_pixe<Pixel, IsPlanar = false, IsStep = false, bool  IsMutable = false> = Pixel*, const version is const Pixel*
// // type_from_x_iterator<Pixel*> = {step_iterator_t = memory_based_step_iterator<Pixel*> xy_locator_t = memory_based_2d_locator<step_iterator_t>}

//     template <typename Pixel>
//     class memory_based_step_iterator 
//     {
//     private:
        
//     };

//     template <typename Pixel>
//     class buffer_view 
//     {
//         using value_type = Pixel;
//         using reference = value_type&;
        
//     };

//     template <typename Pixel>
//     class image_buffer 
//     {
//         using point_t = gil::point_t;

//         template <typename View>
//         image_buffer(View view) 
//         {
//             const auto dimensions = view.dimensions();
//         }

//     private:
//         unsigned char* memory;
//         std::size_t allocated_bytes;
//     };
// }

template <typename View>
__global__ void check_pixels(View view) {
    std::ptrdiff_t x = blockIdx.x * blockDim.x + threadIdx.x;
    std::ptrdiff_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= view.width() || y >= view.height()) {
        return;
    }

    view(x, y) *= 2;
}

int main() {
    std::cout << "hello\n";
    using pixel = gil::rgb8_pixel_t;
    pixel* contents = nullptr;
    std::size_t size = 1920 * 1080;
    hipMalloc(&contents, sizeof(pixel) * size);
    hipMemset(contents, 100, sizeof(pixel) * size);
    auto buffer_view = gil::interleaved_view(1920, 1080, contents, 1920);

    auto deref = [](pixel& p) -> decltype(auto) {
        return p.at(std::integral_constant<int, 0>{});
    };
//    auto view = gil::rgb8_view_t::add_deref<decltype(deref)>::make(buffer_view, deref);
//    std::cout << view(0, 0);
}